#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "./cuda_kernel.cuh"
#include <iostream>
#include <opencv2/core/core.hpp>


const int FILTER_WIDTH = 3;
const int FILTER_HEIGHT = 3;
const int BLOCK_SIZE = 16;

__global__ void totalVarFilter(unsigned char* srcImage, unsigned char* dstImage, unsigned int width, unsigned int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x >= FILTER_WIDTH / 2) && (x < (width - FILTER_WIDTH / 2)) && (y >= FILTER_HEIGHT / 2) && (y < (height - FILTER_HEIGHT / 2)))
    {
        float sod = 0;
        for (int ky = -FILTER_HEIGHT / 2; ky <= FILTER_HEIGHT / 2; ky++) {
            for (int kx = -FILTER_WIDTH / 2; kx <= FILTER_WIDTH / 2; kx++) {
                float fl = srcImage[((y + ky) * width + (x + kx))];
                float center = srcImage[((y)*width + (x))];
                sod += fl - center;
            }
        }
        dstImage[(y * width + x)] = sod;
    }
}



void kernel(unsigned char* srcImage, unsigned char* destImg, unsigned int width, unsigned int height) {
    unsigned char* d_srcImage, *d_dstImage;

    hipMalloc((void**)&d_srcImage, width * height);
    hipMalloc((void**)&d_dstImage, width * height);

    hipMemcpy(d_srcImage, srcImage, width * height, hipMemcpyHostToDevice);
    //hipMemcpy(d_dstImage, srcImage, width * height, hipMemcpyHostToDevice);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    auto kernelT0 = cv::getTickCount();
    totalVarFilter << <grid, block >> > (d_srcImage, d_dstImage, width, height);
    auto kernelDelta = (cv::getTickCount() - kernelT0) / cv::getTickFrequency() * 1000.0000f;
    std::cout << "Time taken by kernel (ms): " << kernelDelta << '\n';

    hipMemcpy(destImg, d_dstImage, width * height, hipMemcpyDeviceToHost);
}
